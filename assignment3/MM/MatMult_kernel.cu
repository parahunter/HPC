#include "hip/hip_runtime.h"
//
// kernel routine
//
#include "AtomicAdd.h"
#include "stdio.h"
void MatMult_gold(const double* A, const double* B, double* C, int M, int N, int K)
//
// Naive version.
//
{

}

extern "C" {
#include <cblas.h>
}

void MatMult_blas(const double* A, const double* B, double* C, int M, int N, int K)
//
// Transposed matrix-vector multiplication using BLAS on CPU
//
{
	cblas_dgemm(CblasRowMajor,CblasNoTrans,CblasNoTrans, M, N, K, 1.0, A, K, B, N, 0.0, C, N);
}

__global__ void MatMult_kernel_v1(const double* A, const double* B, double* C, int M, int N, int K)
//
// Naive version where only global memory and automatic variables are accessed.
//

 // YOUR TASKS:
 // - Write a naive kernel where every thread compute one element of y.
 // - All global memory reads should be coalesced.
 // - Make sure that the kernel does not read or write outside memory allocated.
 //
{
	//reversed order for better coalescence 
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	int index = i * N + j;
	
	//printf("(%d, %d) %d \n", i,j,index);

	double sum = 0.0;

	for(int k = 0; k < K; k++) 
	{
		sum += A[k + i * N] * B[ k*K + j];  
	}

	C[index] = sum;
}


/*
#include "AtomicAdd.h"
__global__ void MatMult_kernel_v2(const double* A, const double* B, double* C, int M, int N, int K)
{
	//reversed order for better coalescence 
	int tid_x = ( blockIdx.x * blockDim.x + threadIdx.x ) * 4;
	int tid_y = ( blockIdx.y * blockDim.y + threadIdx.y ) * 4;
	
	double sm[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

//	for(int i = 0 ; i < 4 * 4 ; i++)				
	//	sm[ i] = 0.0;

	for(int k = 0; k < K; k++) 
	{
		for(int i = 0 ; i < 4 ; i++)
		{	
			for(int j = 0 ; j < 4 ; j++)
			{
				sm[j + 4 * i ] += A[k + (tid_y + i) * K]  * B[ k*N + (tid_x + j)];  
			}
		}
	}

	for(int i = 0 ; i < 4 ; i++)	
	{
		for(int j = 0 ; j < 4 ; j++)
		{
			C[(i+tid_y) * N + (j+tid_x)] = sm[j + 4 * i ];
		}
	}
}
*/

#include "AtomicAdd.h"
__global__ void MatMult_kernel_v2(const double* A, const double* B, double* C, int M, int N, int K)
{
	int blk_x = blockIdx.x * blockDim.x * 4;
	int blk_y = blockIdx.y * blockDim.y * 4;

	//printf("%d,%d,%d,%d\n",blockIdx.x, blockDim.x, blockDim.y, gridDim.x);
	double sm[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};


	for(int k = 0; k < K; k++) 
	{
		for(int i = 0 ; i < 4 ; i++)
		{	
			for(int j = 0 ; j < 4 ; j++)
			{
				int ix = blk_x + i * blockDim.x;
				int ixx = ix + threadIdx.x;
				int iy = blk_y + j * blockDim.y;
				int iyy = iy + threadIdx.y;

					sm[j + 4 * i ] += A[k + iyy * K]  * B[ k*N + ixx];  
				}
	}
	}
	
	
	for(int i = 0 ; i < 4 ; i++)	
	{
		for(int j = 0 ; j < 4 ; j++)
		{

			int ix = blk_x + i * blockDim.x;
			int ixx = ix + threadIdx.x;
			int iy = blk_y + j* blockDim.y;
			int iyy = iy + threadIdx.y;
			C[iyy * N + ixx] = sm[j + 4 * i ];
		}
	}
}

__global__ void MatMult_kernel_v3(const double* A, const double* B, double* C, int M, int N, int K)
{
/*

Suggested steps:
1 - Allocate shared memory
2 - Block the k loop - should still work
3 - Every time I add in steps of k - block shared memory
4 - Change one global memory access to oshared memory access
5 - Finally do the same for B
*/

	int blk_x = blockIdx.x * blockDim.x * 4;
	int blk_y = blockIdx.y * blockDim.y * 4;

	//printf("%d,%d,%d,%d\n",blockIdx.x, blockDim.x, blockDim.y, gridDim.x);
	double sm[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};


	__shared__ double A_s[64][16];
	__shared__ double B_s[16][64];	

	int blockK = 16;
	
	for(int l = 0; l < K/blockK; l++) {
		for(int i = 0; i < 4; i++) {
			int ix = blk_x + l * blockDim.x;
			int ixx = ix + threadIdx.x;
			int iy = blk_y + i * blockDim.y;
			int iyy = iy + threadIdx.y;
		
			A_s[iyy - blk_y][threadIdx.x] = A[ixx + iyy * K];

			ix = blk_x + i * blockDim.x;
			ixx = ix + threadIdx.x;
			iy = blk_y + l * blockDim.y;
			iyy = iy + threadIdx.y;

			B_s[threadIdx.y][ixx - blk_x] = B[ixx + iyy * K];

		}

		__syncthreads();
		for(int kk = 0; kk < blockK; kk++) {
			int k = kk + blockK * l;
			for(int i = 0 ; i < 4 ; i++)
			{	
				for(int j = 0 ; j < 4 ; j++)
				{
					int ix = blk_x + i * blockDim.x;
					int ixx = ix + threadIdx.x;
					int iy = blk_y + j * blockDim.y;
					int iyy = iy + threadIdx.y;

						sm[j + 4 * i ] += A_s[iyy - blk_y][kk] * B[ k*N + ixx];//[k + iyy * K]  * B[ k*N + ixx];  
					}
			}
		}
	}
	
	for(int i = 0 ; i < 4 ; i++)	
	{
		for(int j = 0 ; j < 4 ; j++)
		{

			int ix = blk_x + i * blockDim.x;
			int ixx = ix + threadIdx.x;
			int iy = blk_y + j* blockDim.y;
			int iyy = iy + threadIdx.y;
			C[iyy * N + ixx] = sm[j + 4 * i ];
		}
	}

/*
	//reversed order for better coalescence 
	int tid_x = ( blockIdx.x * blockDim.x + threadIdx.x ) * 4;
	int tid_y = ( blockIdx.y * blockDim.y + threadIdx.y ) * 4;

	__shared__ double A_s[64][16];
	__shared__ double B_s[16][64];	

	

	//printf("%d, %d, %d\n", blockDim.x, gridDim.x, threadIdx.x);

	double sm[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
	int blockK = 16;

	for(int l = 0; l < K/blockK; l++) {

		for(int i = 0 ; i < 4 ; i++) //every thread loads 64*16/256 = 4 elements per A,B
		{	
				A_s[threadIdx.x][threadIdx.y * 4 + i] = A[l * blockK + i + (tid_y) * K];
				B_s[threadIdx.x * 4 + i][threadIdx.y] = B[(l * blockK + i)*N + (tid_x)];	
				
			//		printf("%d, %d\n", threadIdx.x* 4 + i, threadIdx.y);
		}

		__syncthreads();

		for(int kk = 0; kk < blockK ; kk++) 
		{
			int k = kk + blockK * l;
			for(int i = 0 ; i < 4 ; i++)
			{	
				for(int j = 0 ; j < 4 ; j++)
				{
			//		if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) 
						printf("§%f, %f\n", A[k + (tid_y + i) * K], B[ k*N + (tid_x + j)]);  					

					sm[j + 4 * i] += A[k + (tid_y + i) * K]  * B[ k*N + (tid_x + j)];  					
				}
			}
		}

		for(int kk = 0; kk < blockK ; kk++) 
		{
			int k = kk + blockK * l;
			for(int i = 0 ; i < 4 ; i++)
			{	
				for(int j = 0 ; j < 4 ; j++)
				{
			//		if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) 
			//			printf("$%f, %f\n", A_s[i][kk], B_s[kk][j]);  					
				}
			}
		}
		
		//__syncthreads();
	}

	for(int i = 0 ; i < 4 ; i++)	
	{
		for(int j = 0 ; j < 4 ; j++)
		{
			C[(i+tid_y) * N + (j+tid_x)] = sm[j + 4 * i ];
		}
	}
*/
}
extern "C" {
#include <hipblas.h>
}

void MatMult_cublas(const double* d_A, const double* d_B, double* d_C, int M, int N, int K)
//
// Transposed matrix-vector multiplication using CUBLAS on GPU
//
{
	hipblasDgemm('N','N', M, N, K, 1.0, d_B, N, d_A, K, 0.0, d_C, N);
}
